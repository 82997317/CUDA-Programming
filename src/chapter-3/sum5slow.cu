
#include <hip/hip_runtime.h>
#include <math.h> // for abs() function
#include <stdio.h>
#define EPSILON 1.0e-14 // a small number
void __global__ sum(double *x, double *y, double *z, int N);
void check(double *z, int N);

int main(void)
{
    int N = 1024 * 1024;
    int M = sizeof(double) * N;
    double *x = (double*) malloc(M);
    double *y = (double*) malloc(M);
    double *z = (double*) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.0; y[n] = 2.0; z[n] = 0.0;
    }
    double *g_x, *g_y, *g_z;
    hipMalloc((void **)&g_x, M);
    hipMalloc((void **)&g_y, M);
    hipMalloc((void **)&g_z, M);
    hipMemcpy(g_x, x, M, hipMemcpyHostToDevice); 
    hipMemcpy(g_y, y, M, hipMemcpyHostToDevice);
    int block_size = 128;
    int grid_size = N / block_size;
    sum<<<grid_size, block_size>>>(g_x, g_y, g_z, N);
    hipMemcpy(z, g_z, M, hipMemcpyDeviceToHost);
    check(z, N);
    free(x); free(y); free(z);
    hipFree(g_x); hipFree(g_y); hipFree(g_z);
    return 0;
}

void __global__ sum(double *x, double *y, double *z, int N)
{
    int n = gridDim.x * threadIdx.x + blockIdx.x; // correct but bad (slow)
    z[n] = x[n] + y[n];
}

void check(double *z, int N)
{
    int has_error = 0;
    for (int n = 0; n < N; ++n)
    {
        double diff = fabs(z[n] - 3.0);
        if (diff > EPSILON) { has_error = 1; }
    }
    if (has_error) { printf("Has errors.\n"); }
    else { printf("No errors.\n"); }
}
